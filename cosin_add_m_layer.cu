#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cmath>
#include "caffe/layers/cosin_add_m_layer.hpp"


namespace caffe {

  template <typename Dtype>
  __global__ void CosinAddmForward(const int n, const int dim, const Dtype* label,
                                                 Dtype* top_data, Dtype threshold, Dtype bais, Dtype* flag) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      if(top_data[index * dim + gt] < 1.0f) {
          Dtype theta = acos(top_data[index * dim + gt]);
          if (top_data[index * dim + gt] > threshold) {
            top_data[index * dim + gt] = cos(theta + bais);
        }
        else
        {
            top_data[index * dim + gt] = top_data[index * dim + gt] - bais * sin(bais);
            flag[index * dim + gt] = 1.0f;
        }
      }
    }
  }


  template <typename Dtype>
  __global__ void CosinAddmBackward(const int n, const int dim, const Dtype* label,
                                                 Dtype* bottom_diff, const Dtype* cos_data, Dtype bais, const Dtype* flag) {
    CUDA_KERNEL_LOOP(index, n) {
      int gt = static_cast<int>(label[index]);
      if(flag[index * dim + gt] == 0.0f) {
            Dtype cos_theta = cos_data[index * dim + gt];
            Dtype sin_theta = sqrt(1 - pow(cos_theta,2));
            bottom_diff[index * dim + gt] *= cos(bais) + sin(bais) * cos_theta / sin_theta;
        }
    }
  }

  template <typename Dtype>
  void CosinAddmLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                                    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    Dtype* tpflag = top_flag.mutable_gpu_data();
    Dtype* cos_t = cos_theta.mutable_gpu_data();

    int num = bottom[0]->num();
    int count = bottom[0]->count();
    int dim = count / num;

    if (top[0] != bottom[0]) caffe_copy(count, bottom_data, top_data);
    if (!transform_test_ && this->phase_ == TEST) return;

    caffe_copy(count, bottom_data, top_data);
    caffe_copy(count, bottom_data, cos_t);
    caffe_gpu_set(count, Dtype(0), tpflag);

    // NOLINT_NEXT_LINE(whitespace/operators)
    CosinAddmForward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
      num, dim, label_data, top_data, threshold, m_, tpflag);
    CUDA_POST_KERNEL_CHECK;

    const Dtype* cos_test = cos_theta.cpu_data();
    const Dtype* tpflag_test = top_flag.cpu_data();
    const Dtype* top_data_test = top[0]->cpu_data();
  }

  template <typename Dtype>
  void CosinAddmLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                     const vector<bool>& propagate_down,
                                                     const vector<Blob<Dtype>*>& bottom) {
    if (top[0] != bottom[0] && propagate_down[0]) {

      int num = bottom[0]->num();
      int count = bottom[0]->count();
      int dim = count / num;

      const Dtype* top_diff = top[0]->gpu_diff();
      const Dtype* label_data = bottom[1]->gpu_data();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

      caffe_copy(count, top_diff, bottom_diff);

      const Dtype* tpflag = top_flag.gpu_data();
      const Dtype* cos_t = cos_theta.gpu_data();

      CosinAddmBackward<Dtype> << <CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS >> > (
        num, dim, label_data, bottom_diff, cos_t, m_, tpflag);
      CUDA_POST_KERNEL_CHECK;

    }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(CosinAddmLayer);
}  // namespace caffe
